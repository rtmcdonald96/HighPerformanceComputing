
#include <hip/hip_runtime.h>
#include <iostream>
/*
Ryan McDonald
CSUF Spring 2021
CPSC 479 - Dr. Bein
*/
__global__
void setVal(int n, int* x) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		x[i]=0;
}

__global__
void addVal(int n, int* x){

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		x[i] +=i;
}

int main(int argc, char* argv[])
{
	//Set size of array
	int N = 1024;
	int* x;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(int));

	//Run Kernel on 1024 elements on the GPU
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	
	setVal<<<numBlocks, blockSize>>>(N, x);	
	addVal<<<numBlocks, blockSize>>>(N, x);

	//Waits for GPU to finish before accessing data on the host
	hipDeviceSynchronize();

	//Optional code to show value at each index. 
	//Used to confirm setVal and addVal function worked.
	/*
	for (int i=0; i<N; i++)
	{
	std::cout << "Index x[" << i << "] has value: "<< x[i] << std::endl;
	}
	*/

	//Free memory
	hipFree(x);

	return 0;
}
