
#include <hip/hip_runtime.h>
#include <iostream>
/*
Ryan McDonald
CSUF Spring 2021
CPSC 479 - Dr. Bein
*/
__global__
void setVal(int n, int* x) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		x[i]=0;
}

int main(int argc, char* argv[])
{
	//Set size of array
	int N = 16;
	int* x;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(int));

	//Run Kernel on 16 elements on the GPU
	int blockSize = 4;
	int numBlocks = (N + blockSize - 1) / blockSize;

	setVal<<<numBlocks, blockSize>>>(N, x);	

	//Waits for GPU to finish before accessing data on the host
	hipDeviceSynchronize();
	
	//Optional code to show value at each index. 
	//Used to confirm setVal function worked.
	/*
	for (int i=0; i<N; i++)
	{
	std::cout << "Index x[" << i << "] has value: "<< x[i] << std::endl;
	}
	*/

	//Free memory
	hipFree(x);

	return 0;
}