
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
/*
Ryan McDonald
CSUF Spring 2021
CPSC 479 - Dr. Bein
*/

__global__ 
void squareMatrix(int* matrix, int* result, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;

	if (row < N && col < N)
	{
		for (int i = 0; i < N; i++)
		{
			sum += matrix[row * N + i] * matrix[i * N + col];
		}
		result[row * N + col] = sum;
	}
}

__global__ void findLowestVal(int* matrix, int* lowestVal, int N) {

	__shared__ int c[256];

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int offset = 0;
	int temporary = matrix[0];

	while (index + offset < N)
	{
		temporary = min(temporary, matrix[index + offset]);
		offset += stride;
	}
	c[threadIdx.x] = temporary;

	 int u = blockDim.x / 2;
	while (u != 0) {
		if (threadIdx.x < u) {
			c[threadIdx.x] = min(c[threadIdx.x], c[threadIdx.x + u]);
		}
		__syncthreads();
		u = u/2;
	}

	if (threadIdx.x == 0)
	{
		*lowestVal = min(*lowestVal, c[0]);
	}

}

//Function to fill matrix with random values
void fill_matrix(int* matrix, int N) {
	for (int i = 0; i < N; i++)
	{
		matrix[i] = rand() % 100;
	}
}

int main(int argc, char* argv[]) {
	//Set size of matrix [16x16] Square Matrix
	int N = 16 * 16;
	int* myMatrix;
	int* myMatrixSquared;
	int* lowestVal;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&lowestVal, sizeof(int));
	hipMallocManaged(&myMatrix, N * sizeof(int));
	hipMallocManaged(&myMatrixSquared, N * sizeof(int));

	//Populate Matrix with random values
	fill_matrix(myMatrix, N);

	int blockSize = 16;
	int numBlocks = (N + blockSize - 1) / blockSize;

	findLowestVal <<<numBlocks, blockSize >>> (myMatrix, lowestVal, N);

	dim3 threadsPerBlock(blockSize, blockSize);
	dim3 blocksPerGrid(numBlocks, numBlocks);

	squareMatrix<<<blocksPerGrid, threadsPerBlock >>> (myMatrix, myMatrixSquared, N);

	//Waits for GPU to finish before accessing data on the host
	hipDeviceSynchronize();

	//Free memory
	hipFree(lowestVal);
	hipFree(myMatrix);
	hipFree(myMatrixSquared);

	return 0;
}